
#include <hip/hip_runtime.h>
/*     CalculiX - A 3-dimensional finite element program                 */
/*              Copyright (C) 1998-2011 Guido Dhondt                     */
/*     This subroutine                                                   */
/*              Copyright (C) 2013 Peter A. Gustafson                    */
/*                                                                       */
/*     This program is free software; you can redistribute it and/or     */
/*     modify it under the terms of the GNU General Public License as    */
/*     published by the Free Software Foundation(version 2);    */
/*                                                                       */

/*     This program is distributed in the hope that it will be useful,   */
/*     but WITHOUT ANY WARRANTY; without even the implied warranty of    */ 
/*     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the      */
/*     GNU General Public License for more details.                      */

/*     You should have received a copy of the GNU General Public License */
/*     along with this program; if not, write to the Free Software       */
/*     Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA.         */

#ifdef CUDACUSP

#include <cusp/hyb_matrix.h>
#include <cusp/dia_matrix.h>
// #include <cusp/gallery/poisson.h>
#include <cusp/krylov/cg.h>
// #include <cusp/krylov/cg_m.h>
// #include <cusp/krylov/bicg.h>
// #include <cusp/krylov/bicgstab.h>
#include <cusp/version.h>
// #include <cusp/print.h>
#include <cusp/array1d.h>
#include <cusp/multiply.h>
#include <cusp/precond/ainv.h> 
#include <iostream>
#include <cusp/precond/smoothed_aggregation.h>
// #include <cusp/krylov/gmres.h>
// #include <cusp/detail/format_utils.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
// #include <cusp/ell_matrix.h>


template <typename Monitor>
void report_status(Monitor& monitor)
{
  if (monitor.converged())
    {
      std::cout << "  Solver converged to " << monitor.tolerance() << " tolerance";
      std::cout << " after " << monitor.iteration_count() << " iterations";
      std::cout << " (" << monitor.residual_norm() << " final residual)" << "\n";
    }
  else
    {
      std::cout << "  Solver reached iteration limit " << monitor.iteration_limit() << " before converging";
      std::cout << " to " << monitor.tolerance() << " tolerance ";
      std::cout << " (" << monitor.residual_norm() << " final residual)" << "\n";
    }
  std::cout <<  "\n\n";
}


// which floating point type to use
typedef double ValueType;
// typedef cusp::host_memory MemorySpace;
typedef cusp::device_memory MemorySpace;
// int global_recalc_cuda_M = 1;
// Can create pointers to precond matrices... can't transfer pointers to device and back as of 7/17/2013
// cusp::precond::bridson_ainv<ValueType, MemorySpace> *MM;
// cusp::precond::bridson_ainv<ValueType, cusp::host_memory> *M;




template <typename T>
struct invsqr : public thrust::unary_function<T,T>
{
  __host__ __device__
  T operator()(const T& v) 
  {
    return T (1.0)/sqrt(v);
  }
};

template <typename T>
struct absolute : public thrust::unary_function<T,T>
{
    __host__ __device__
    T operator()(T x)
  {
    return x < 0 ? -x : x;
  }
};

extern "C"
int cudacusp(double *ad, double *au, double *adb, double *aub, double *sigma, 
	     double *b, int *icol, int *irow, int *neq, int *nzs, 
	     int *symmetryflag, int *inputformat, int *jq, int *nzs3)
{
  int cuda_major =  CUDA_VERSION / 1000;
  int cuda_minor = (CUDA_VERSION % 1000) / 10;

  int thrust_major = THRUST_MAJOR_VERSION;
  int thrust_minor = THRUST_MINOR_VERSION;

  int cusp_major = CUSP_MAJOR_VERSION;
  int cusp_minor = CUSP_MINOR_VERSION;

  clock_t timeb;
  clock_t timee;

  std::cout << " Using CUDA based on CUSP CG SOLVER\n";
  std::cout << "   CUDA   v" << cuda_major   << "." << cuda_minor   << "\n";
  std::cout << "   Thrust v" << thrust_major << "." << thrust_minor << "\n";
  std::cout << "   Cusp   v" << cusp_major   << "." << cusp_minor   << "\n";


  timeb = clock();
  /* Fill the matrix.  
     The off diagonal triangle is columnar from ccx
     irow() identifies the row within the column
     icol() identifies the number of non zeros within the column
     Move the the next column after achieving icol() within a column. /*

     Since cusp need be row sorted, we enter the transpose */
  // Create the row and column indices

  int i,j,k,l,m,n;
  int nvals=0;

  // Test for non zero values
  for (i=0; i<*neq; i++){if (ad[i]<0) nvals++;}
  if (nvals) {thrust::transform(ad, ad+*neq, ad, absolute<ValueType>());}
  
  k=0; // data index
  l=0; // row index
  m=0; // column tracker index

  // ASSEMBLE UPPER ONLY // cusp::coo_matrix<int, ValueType, cusp::host_memory> AU(*neq,*neq,*nzs);
  // ASSEMBLE UPPER ONLY // for (i = 0; i < *neq; i++){
  // ASSEMBLE UPPER ONLY //   for (j = 0; j < icol[i]; j++){
  // ASSEMBLE UPPER ONLY //     n = irow[m]-1;
  // ASSEMBLE UPPER ONLY //     AU.row_indices[k] = l; 
  // ASSEMBLE UPPER ONLY //     AU.column_indices[k] = n; 
  // ASSEMBLE UPPER ONLY //     AU.values[k++] = au[m++];
  // ASSEMBLE UPPER ONLY //   }
  // ASSEMBLE UPPER ONLY //   l++;
  // ASSEMBLE UPPER ONLY // }
  // ASSEMBLE UPPER ONLY // timee = clock();
  // ASSEMBLE UPPER ONLY // std::cout << "  Assemble upper triangular time = " << 
  // ASSEMBLE UPPER ONLY //   (double(timee)-double(timeb))/double(CLOCKS_PER_SEC) << "\n\n";

  cusp::coo_matrix<int, ValueType, cusp::host_memory> A(*neq,*neq,2*(*nzs)+*neq);
  // ASSEMBLE FULL MATRIX //
  for (i = 0; i < *neq; i++){
    A.row_indices[k] = i; 
    A.column_indices[k] = i; 
    A.values[k++] = ad[i];
    for (j = 0; j < icol[i]; j++){
      n = irow[m]-1;
      A.row_indices[k] = l; 
      A.column_indices[k] = n; 
      A.values[k++] = au[m];
      A.row_indices[k] = n; 
      A.column_indices[k] = l; 
      A.values[k++] = au[m++];
    }
    l++;
  }

  // cusp::print(A);
  A.sort_by_row_and_column();
  // cusp::print(A);
  cusp::hyb_matrix<int, ValueType, MemorySpace> AA = A;
  timee = clock();
  std::cout << "  Assembled stiffness matrix on CUDA device in = " << 
    (double(timee)-double(timeb))/double(CLOCKS_PER_SEC) << "\n\n";

  timee = clock();
  // CONVERT UPPER TO FULL MATRIX
  //
  // Start on device version
  // ON DEVICE // cusp::hyb_matrix<int, ValueType, MemorySpace> AA = AU;
  // ON DEVICE // // Bring the matrices together limiting scope as much as possible
  // ON DEVICE // {
  // ON DEVICE //   cusp::hyb_matrix<int, ValueType, MemorySpace> AAT;
  // ON DEVICE //   cusp::transpose(AA,AAT);
  // ON DEVICE //   cusp::add(AA,AAT,AA);
  // ON DEVICE // } // free AAT
  // ON DEVICE // {
  // ON DEVICE //   cusp::coo_matrix<int, ValueType, MemorySpace> DD(*neq,*neq,*neq);
  // ON DEVICE //   // Potentially not the most efficient possible
  // ON DEVICE //   thrust::sequence (DD.row_indices.begin(),DD.row_indices.end());
  // ON DEVICE //   thrust::sequence (DD.column_indices.begin(),DD.column_indices.end());
  // ON DEVICE //   thrust::copy (ad, ad+*neq, DD.values.begin());
  // ON DEVICE //   cusp::add(AA,DD,AA);
  // ON DEVICE // }
  // End on device version
  // Start on host version
  // ON HOST // {
  // ON HOST //   cusp::hyb_matrix<int, ValueType, cusp::host_memory> AAT;
  // ON HOST //   cusp::transpose(AU,AAT);
  // ON HOST //   cusp::add(AU,AAT,AU);
  // ON HOST // } // free AAT
  // ON HOST // {
  // ON HOST //   cusp::coo_matrix<int, ValueType, cusp::host_memory> DD(*neq,*neq,*neq);
  // ON HOST //   // Potentially not the most efficient possible
  // ON HOST //   thrust::sequence (DD.row_indices.begin(),DD.row_indices.end());
  // ON HOST //   thrust::sequence (DD.column_indices.begin(),DD.column_indices.end());
  // ON HOST //   thrust::copy (ad, ad+*neq, DD.values.begin());
  // ON HOST //   cusp::add(AU,DD,AU);
  // ON HOST // }
  // ON HOST // cusp::hyb_matrix<int, ValueType, MemorySpace> AA = AU;
  // End on host version

  // timee = clock();
  // std::cout << "  Time to assemble AA = " << 
  //   (double(timee)-double(timeb))/double(CLOCKS_PER_SEC) << "\n\n";
  
  // cusp::hyb_matrix<int, ValueType, cusp::host_memory> M;
  // cusp::hyb_matrix<int, ValueType, MemorySpace> MMM;
  // if (inccholpre) {
  //   int ier;
  //   AT = PreConditionCudaCusp (A,neq,nzs,&ier);  // Lower triangular, reuse AT
  //   // printf ("A");
  //   // cusp::print(A);
  //   cusp::transpose(AT,AU);
  //   cusp::multiply(AT,AU,M);
  //   
  //   // printf ("M");
  //   // cusp::print(M);
  //   MMM=M;
  // } 
  
  timeb = clock();
  // set preconditioners
  // cusp::identity_operator<ValueType, MemorySpace> MM(A.num_rows, A.num_rows);
  // AINV preconditioner, using standard drop tolerance strategy 
  // cusp::precond::scaled_bridson_ainv<ValueType, MemorySpace> MM(AA, .1);
  // printf ("Scaled bridson with .1 drop tolerarance\n");
  // cusp::precond::scaled_bridson_ainv<ValueType, MemorySpace> MM(AA, .1);
  // int nunsc = 15;
  // printf ("Scaled bridson with %i non-zeros per row\n", nunsc);
  // cusp::precond::scaled_bridson_ainv<ValueType, MemorySpace> MM(AA, 0, nunsc);
  // printf ("Unscaled bridson with %i non-zeros per row\n", nunsc);
  // cusp::precond::bridson_ainv<ValueType, MemorySpace> MM(AA, 0, nunsc);

  // The compiler warns about race conditions with smoothed.
  // printf ("Smoothed aggregation on device\n");
  // cusp::precond::smoothed_aggregation<int, ValueType, MemorySpace> MM(AA);
  printf ("Diagnonal preconditioner\n");
  cusp::precond::diagonal<ValueType, MemorySpace> MM(AA);
  timee = clock();
  std::cout << "  Preconditioning time = " << 
    (double(timee)-double(timeb))/double(CLOCKS_PER_SEC) << "\n\n";
  
  // allocate storage for and copy right hand side (BB). 
  cusp::array1d<ValueType, MemorySpace> BB(*neq, 0.0);
  thrust::copy (b, b+*neq, BB.begin());
  
  timeb = clock();
  // set stopping criteria 
  // http://docs.cusp-library.googlecode.com/hg/classcusp_1_1default__monitor.html
  // ||b - A x|| <= absolute_tolerance + relative_tolerance * ||b||
  
  i=50000;
  // if ((*b)<0.0){
  if (nvals){
    // Non-positive definite.  Give up quickly after spawning an answer
    // thrust::copy (ad, ad+*neq, DD.begin());
    // thrust::transform(DD.begin(), DD.end(), DD.begin(), absolute<ValueType>());
    i=0;
    printf ("There are %i negative values on the diagonal.  The attempt is abandoned.\n", nvals);
  }
  cusp::verbose_monitor<ValueType> monitor(BB, i, 1e-6);

  // solve the linear system AA * XX = BB 
  cusp::krylov::cg(AA, BB, BB, monitor, MM); //Conjugate Gradient method
  // cusp::krylov::cg(AA, BB, BB, monitor); //Conjugate Gradient method
  // cusp::krylov::bicgstab(AA, BB, BB, monitor, MM); //BiConjugate Gradient Stabilized method
  // cusp::krylov::bicg(AA, AA, BB, BB, monitor, MM, MM); //BiConjugate Gradient
  
  // solve the linear system AA * XX = BB with the GMRES.
  // Cost grows as O(n^2) with n being number of iterations.  Thus,
  // can restart with a fraction of the answer as an initial guess
  // thrust::fill( BB.begin(), BB.end(), ValueType(0) );
  // int restart = 20;
  // timeb = clock();
  // cusp::krylov::gmres(AA, BB, BB, restart, monitor, MM);
  // cusp::krylov::gmres(AA, BB, BB, restart, monitor);
  // timee = clock();
  
  timee = clock();

  std::cout << "  CUDA iterative solver time = " << 
    (double(timee)-double(timeb))/double(CLOCKS_PER_SEC) << "\n\n";

  // report status
  // report_status(monitor);
  
  // Works only with smoothed_aggregation
  // std::cout << "\nPreconditioner statistics" << "\n";
  // M.print();
  
  // Copy the result to the b array
  thrust::copy (BB.begin(), BB.end(), b);

  if (!monitor.converged()){
    printf (" WARNING: Cuda Cusp did not find a solution.\n");
  }
  return 0;
}
#endif

